
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

// Define constants and data types
#define PAGE_SIZE           32
#define PHYSICAL_MEM_SIZE   32768
#define STORAGE_SIZE        131072
#define DATAFILE            "./data.bin"
#define OUTPUTFILE          "./snapshot.bin"
typedef unsigned char uchar;
typedef uint32_t u32;
const uint32_t VALID    		= 0 | 1;
const uint32_t INVALID			= 0;
const uint32_t PAGENUMBERMASK	= 0x00003FFE;
const uint32_t LASTTIMEMASK		= 0xFFFFC000;
const uint32_t DNE				= 0xFFFFFFFF;

// Declare variables
__device__ __managed__ int PAGE_ENTRIES = 0;
__device__ __managed__ int PAGEFAULT = 0;
__device__ __managed__ int CURRENTTIME = 0;
__device__ __managed__ u32 LRU[PHYSICAL_MEM_SIZE/PAGE_SIZE];
__device__ __managed__ uchar storage[STORAGE_SIZE];
__device__ __managed__ uchar results[STORAGE_SIZE];
__device__ __managed__ uchar input[STORAGE_SIZE];
extern __shared__ u32 pageTable[];

// Function
// ******************************************************************
// Initialize
__device__ void initPageTable(int entries) {
    for (int i = 0; i < entries; i++) {
        pageTable[i] = DNE;
		LRU[i] = 0;
    }
}
// ******************************************************************

// ******************************************************************
// File I/O
int loadBinaryFile(char *fileName, uchar *input, int storageSize) {
    FILE *fptr = fopen(fileName, "rb");
    // Get size
    fseek(fptr, 0, SEEK_END);
    int size = ftell(fptr);
    rewind(fptr);
    // Read data from input file
    fread(input, sizeof(unsigned char), size, fptr);
    if (storageSize < size) {
        printf("ERROR: Storage size is too small to store input data!\n");
    }
	fclose(fptr);
    return size;
}

void writeBinaryFile(char *fileName, uchar *input, int storageSize) {
    FILE *fptr = fopen(fileName, "wb");
    // Read data from input file
    fwrite(input, sizeof(unsigned char), storageSize, fptr);
	fclose(fptr);
}
// ******************************************************************

// ******************************************************************
// Read/Write
__device__ u32 isValid(u32 PTE) {
	return PTE & VALID;
}
__device__ u32 getPageNumber(u32 PTE) {
	return (PTE & PAGENUMBERMASK) >> 1;
}
__device__ u32 getLastUsedTime(u32 PTE) {
	return (PTE & LASTTIMEMASK) >> 14;
}
__device__ u32 makePTE(u32 time, u32 pageNumber, u32 validbit) {
	return (time << 14) | (pageNumber << 1) | validbit;
}
__device__ u32 paging(uchar *memory, u32 pageNumber, u32 pageOffset) {
	CURRENTTIME++;

	// Find if the target page exists
	for (u32 i = 0; i < PAGE_ENTRIES; i++) {
		if (pageTable[i] == pageNumber) {
			// Update time
			LRU[i] = CURRENTTIME;
			return i * PAGE_SIZE + pageOffset;
		}
	}

	// Find if there is a empty entry to place
	for (u32 i = 0; i < PAGE_ENTRIES; i++) {
		if (pageTable[i] == DNE) {
			// Because of a empty hole, it must be a pagefault
			PAGEFAULT++;
			// Update PTE
			pageTable[i] = pageNumber;
			LRU[i] = CURRENTTIME;
			return i * PAGE_SIZE + pageOffset;
		}
	}

	// Find a place for swaping in by the RULE of LRU
	u32 leastEntry = DNE;
	u32 leastTime  = DNE;
	for (u32 i = 0; i < PAGE_ENTRIES; i++) {
		if (leastTime > LRU[i]) {
			leastTime = LRU[i];
			leastEntry = i;
		}
	}
	// Replace & update infos
	PAGEFAULT++;
	pageTable[leastEntry] = pageNumber;
	LRU[leastEntry] = CURRENTTIME;
	return leastEntry * PAGE_SIZE + pageOffset;
}

__device__ uchar Gread(uchar *memory, u32 address) {
	u32 pageNumber = address/PAGE_SIZE;
	u32 pageOffset = address%PAGE_SIZE;

	u32 reMappingAddress = paging(memory, pageNumber, pageOffset);
	return memory[reMappingAddress];
}

__device__ void Gwrite(uchar *memory, u32 address, uchar writeValue) {
	u32 pageNumber = address/PAGE_SIZE;
	u32 pageOffset = address%PAGE_SIZE;

	u32 reMappingAddress = paging(memory, pageNumber, pageOffset);
	memory[reMappingAddress] = writeValue;
}

__device__ void snapshot(uchar *result, uchar *memory, int offset, int input_size) {
	for (int i = 0; i < input_size; i++) {
		result[i] = Gread(memory, i+offset);
	}
}
// ******************************************************************

// ******************************************************************
// Kernel function
__global__ void mykernel(int input_size) {
    __shared__ uchar data[PHYSICAL_MEM_SIZE];
    PAGE_ENTRIES = PHYSICAL_MEM_SIZE/PAGE_SIZE;
    initPageTable(PAGE_ENTRIES);
	//##Gwrite / Gread code section start###
	for(int i = 0; i < input_size; i++) {
		Gwrite(data, i, input[i]);
	}
	for(int i = input_size - 1; i >= input_size - 10; i--) {
		int value = Gread(data, i);
	}
	//the last line of Gwrite/Gread code section should be snapshot ()
	snapshot(results, data, 0, input_size);
	//###Gwrite/Gread code section end### 
    printf("pagefault times = %d\n", PAGEFAULT);
}
// ******************************************************************

int main() {
    int input_size = loadBinaryFile(DATAFILE, input, STORAGE_SIZE);
	printf("Loading...\n");
    //cudaSetDevice(2);
    mykernel<<<1, 1, 16384>>>(input_size);
    hipDeviceSynchronize();
    hipDeviceReset();

    writeBinaryFile(OUTPUTFILE, results, input_size);
    return 0;
}
